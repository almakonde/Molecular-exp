#include "hip/hip_runtime.h"
﻿#include "SolverGPU.cuh"




void check(hipError_t x) {
    fprintf(stderr, "%s\n", hipGetErrorString(x));
}


typedef unsigned long uint64_t;

void PrintCudaCards()
{
	int  GPU_N;

	printf("Starting MultiGPU\n");
	hipGetDeviceCount(&GPU_N);

	if (GPU_N > 4)
	{
		GPU_N = 4;
	}

	printf("CUDA-capable device count: %i\n", GPU_N);

}

__global__ void initkernelParticals(Player *bots,int numElements, GameConstants* gconst)
{
    int  tid = blockIdx.x * blockDim.x + threadIdx.x;
    int  threadN = gridDim.x * blockDim.x;


    for (int pos = tid; pos < numElements; pos += threadN)
    {
        for (int l=0;l<gconst->TypeSize;l++)
        {
            bots[pos].typeAmount[l]=0;
            for (int l1=0;l1<gconst->TypeSize;l1++)
            {
                bots[pos].typeAmountPP[l][l1]=0;
            }
        }
    }
}

__device__ inline float cLength(Vec3 v)
{
    float lSq = cLengthSq(v);
    if (lSq>0.0f)
        return  sqrtf(lSq);
    else
        return 0.0f;
}


__device__ inline float cLengthSq(Vec3 v)
{
    return Dot(v,v);
}

__device__ inline Vec3 cSafeNormalize(Vec3 v, Vec3 fallback)
{
    float l = cLengthSq(v);
    if (l > 0.0f)
    {
        return v * cInvSqrt(l);
    }
    else
        return fallback;
}

__device__ inline float cDistance(Vec3 v1, Vec3 v2)
{
    return cLength(v1-v2);
}

__device__ inline float cInvSqrt(float x)
{
    return 1.0f/sqrtf(x);
}


__device__ Vec3 projectUonV(const Vec3& u, const Vec3& v)
{
    Vec3 r=Vec3(0,0,0);
    float DotVV=Dot(v, v);
    if(DotVV!=0)
        r = v* (Dot(u, v)/DotVV);
    return r;
}

__device__ void performCollision(Player& s1, Player& s2)
{

        float softCoeff=0.973f;
        Vec3 nv1; // new velocity for sphere 1
        Vec3 nv2; // new velocity for sphere 2
        // this can probably be optimised a bit, but it basically swaps the velocity amounts
        // that are perpendicular to the surface of the collistion.
        // If the spheres had different masses, then u would need to scale the amounts of
        // velocities exchanged inversely proportional to their masses.
        nv1 = s1.velocity;
        nv1 =nv1+ projectUonV(s2.velocity, s2.currPos- s1.currPos);
        nv1 =nv1- projectUonV(s1.velocity, s1.currPos- s2.currPos);
        nv2 = s2.velocity;
        nv2 =nv2+ projectUonV(s1.velocity, s2.currPos- s1.currPos);
        nv2 =nv2- projectUonV(s2.velocity, s1.currPos- s2.currPos);
        s1.velocity = softCoeff*nv1;
        s2.velocity = softCoeff*nv2;
}



__device__ inline int array2D(int a, int b, int numElements)
{
    return a*numElements+b;
}
__global__ void UpdateBallVelocity(Player* bot, Constraint* allconstr, GameConstants* gconst,int numElements)
{
    int  tid = blockIdx.x * blockDim.x + threadIdx.x;
    int  threadN = gridDim.x * blockDim.x;

    float m=10.0f;
    float deltaTime=gconst->deltaTime;


    for (int pos = tid; pos < numElements; pos += threadN)
    {
        Vec3 directForce=Vec3(0,0,0);
        for (int i = 0; i< numElements; i++)
        {
          if(pos!=i)
          {
               bool collide=false;
               bot[pos].force=0.0f;
               float dotV2=cDistance(bot[pos].currPos,bot[i].currPos);
               float dotV=cDistance(bot[pos].currPos,bot[i].currPos)-(bot[pos].radius+bot[i].radius);
               float dotVC=cDistance(bot[pos].currPos,Vec3(0,0,0));
               Vec3 directPP=bot[pos].currPos-bot[i].currPos;


               if(dotV<0.0f)
               {
                   collide=true;
               }

              if(dotV<15.0f)
              // if(bot[pos].link)
               {
                   if(LinksField[bot[pos].type][bot[i].type]==1)
                       bot[pos].force-=directPP*gconst->gravconst2*(bot[pos].mass*bot[i].mass)/(dotV2*dotV2+0.001f);
                   else {
                       bot[pos].force+=directPP*gconst->gravconst2*(bot[pos].mass*bot[i].mass)/(dotV2*dotV2+0.001f);
                   }
               }

               if (allconstr[array2D(pos,i,numElements)].active )
                       //or allconstr[array2D(i,pos,numElements)].active)
               {

                   if (dotV>gconst->constrLenth)
                        //bot[pos].force-=directPP*gconst->gravconst*(bot[pos].mass*bot[i].mass)/(1-(dotV2-gconst->constrLenth)/gconst->constrLenth);
                       bot[pos].force-=directPP*gconst->gravconst*(bot[pos].mass*bot[i].mass)/abs(1-(dotV-gconst->constrLenth)/gconst->constrLenth);
                   if(dotV<gconst->constrLenth)
                        //bot[pos].force+=directPP*gconst->gravconst*(bot[pos].mass*bot[i].mass)/(1-(gconst->constrLenth-dotV2)/gconst->constrLenth);
                       bot[pos].force+=directPP*gconst->gravconst*(bot[pos].mass*bot[i].mass)/abs(1-(gconst->constrLenth-dotV)/gconst->constrLenth);



               }
               else
               {

                   if(dotVC>175)
                   {
                        bot[pos].force-=directPP*gconst->gravconst*(bot[pos].mass*bot[pos].mass)/(dotVC*dotVC+0.001f);
                        //bot[pos].velocity=bot[pos].velocity+deltaTime*bot[pos].force;
                   }

               }

               bot[pos].velocity=bot[pos].velocity+deltaTime*bot[pos].force;


               if(collide)
               {
                   //move to real point of collide
                   float dotV2=cDistance(bot[pos].currPos,bot[i].currPos);
                   float dist=0.001+(bot[pos].radius+bot[i].radius)-dotV2;
                   float distHalf=dist/2.f;
                   Vec3 directPP=bot[pos].currPos-bot[i].currPos;
                   bot[pos].currPos=bot[pos].currPos+ distHalf*cSafeNormalize(directPP,Vec3(0,0,0));
                   bot[i].currPos=bot[i].currPos- distHalf*cSafeNormalize(directPP,Vec3(0,0,0));

                   if(!bot[pos].collide)
                   {
                     bot[pos].collide=true;
                     bot[i].collide=true;
                     performCollision(bot[pos],bot[i]);
                     bot[i].force=Vec3(0,0,0);
                     bot[pos].force=Vec3(0,0,0);
                   }
               }

            }
        }



    }
}


__global__ void UpdateBallPos(Player* bot, GameConstants* gconst, int numElements)
{
    int  tid = blockIdx.x * blockDim.x + threadIdx.x;
    int  threadN = gridDim.x * blockDim.x;

    float deltaTime=gconst->deltaTime;

    for (int pos = tid; pos < numElements; pos += threadN)
    {
        //float dotV=cDistance(bot[pos].currPos,Vec3(0,0,0));
       // if(dotV>95.0f)
       //     bot[pos].velocity=-0.75f*bot[pos].velocity;

        if(cLength(bot[pos].velocity)>gconst->speedLimit)
        {
            bot[pos].velocity=cSafeNormalize(bot[pos].velocity, Vec3(0,0,0))*gconst->speedLimit;
        }
/*
        int sumlink=0;
        for(int l=0;l<3;l++)
        {
            sumlink+=bot[pos].typeAmount[l];
        }

        if(sumlink>0)
            bot[pos].link=true;
        else {
            bot[pos].link=false;
        }

        */

        //if(abs(bot[pos].currPos.x)>gconst->WIDTH)
        //    bot[pos].velocity.x=-bot[pos].velocity.x;
        //if(abs(bot[pos].currPos.y)>gconst->HEIGHT)
        //    bot[pos].velocity.y=-bot[pos].velocity.y;
        //if(abs(bot[pos].currPos.z)>gconst->DEPTH)
        //    bot[pos].velocity.z=-bot[pos].velocity.z;

        bot[pos].currPos=bot[pos].currPos+deltaTime*bot[pos].velocity;
        bot[pos].collide=false;

        bot[pos].velocity_old=bot[pos].velocity;

    }
}

__global__ void UpdateBallPos2(Player* bot, int numElements)
{
    int  tid = blockIdx.x * blockDim.x + threadIdx.x;
    int  threadN = gridDim.x * blockDim.x;

    float m=10.0f;
    float deltaTime=0.01f;

    for (int pos = tid; pos < numElements; pos += threadN)
    {
        //float dotV=cDistance(bot[pos].currPos,Vec3(0,0,0));
       // if(dotV>95.0f)
       //     bot[pos].velocity=-0.75f*bot[pos].velocity;
       // bot[pos].accelerate=bot[pos].force/bot[pos].mass;
        bot[pos].velocity=bot[pos].velocity+deltaTime*bot[pos].accelerate;
        bot[pos].currPos=bot[pos].currPos+deltaTime*bot[pos].velocity;
        bot[pos].collide=false;
        bot[pos].velocity_old=bot[pos].velocity;
        bot[pos].force=Vec3(0, 0, 0);
    }
}

__global__ void UpdateConstrains2(Player* bot, Constraint* constr, Constraint* allconstr, GameConstants* gconst, int numElements, int numElements2)
{
    int  tid = blockIdx.x * blockDim.x + threadIdx.x;
    int  threadN = gridDim.x * blockDim.x;

    float m=10.0f;
    float deltaTime=gconst->deltaTime;

    for (int pos = tid; pos < numElements; pos += threadN)
    {
        for (int i = 0; i< numElements; i++)
        {
          if(pos!=i)
          {
               float dotV=cDistance(bot[pos].currPos,bot[i].currPos)-(bot[pos].radius+bot[i].radius);
               allconstr[array2D(pos,i, gconst->number_of_bots)].restlength=dotV;


               if(dotV<gconst->constrLenthCreate && Links[bot[pos].type][bot[i].type]==1
                      // && bot[pos].typeAmount[bot[i].type]<LinkTypeSize[bot[i].type]
                        && bot[pos].typeAmount[bot[i].type]<LinkTypeSize[bot[i].type]
                        && bot[i].typeAmount[bot[pos].type]<LinkTypeSize[bot[pos].type]
                       && bot[pos].typeAmountPP[bot[pos].type][bot[i].type]<LinkTypePP[bot[pos].type][bot[i].type]
                       && bot[i].typeAmountPP[bot[i].type][bot[pos].type]<LinkTypePP[bot[i].type][bot[pos].type]
                       && (allconstr[array2D(pos,i, gconst->number_of_bots)].active!=true or
                           allconstr[array2D(i,pos, gconst->number_of_bots)].active!=true)
                       //&& bot[i].typeAmount[bot[pos].type]<LinkTypeSize[bot[pos].type]
                       )
               {
                   //constr[pos].active=true;
                  // atomicAdd(&bot[pos].typeAmount[bot[i].type],1);
                  // atomicAdd(&bot[i].typeAmount[bot[pos].type],1);
                   bot[pos].typeAmount[bot[i].type]+=1;
                   bot[i].typeAmount[bot[pos].type]+=1;
                   bot[pos].typeAmountPP[bot[pos].type][bot[i].type]+=1;
                   bot[i].typeAmountPP[bot[i].type][bot[pos].type]+=1;
                   allconstr[array2D(pos,i, gconst->number_of_bots)].active=true;
                   allconstr[array2D(i,pos, gconst->number_of_bots)].active=true;

                   bot[pos].link=true;
                   bot[i].link=true;

               }


           }
        }
    }
}
__global__ void UpdateConstrains3(Player* bot, Constraint* constr, Constraint* allconstr, GameConstants* gconst, int numElements, int numElements2)
{
        int  tid = blockIdx.x * blockDim.x + threadIdx.x;
        int  threadN = gridDim.x * blockDim.x;

        float m=10.0f;
        float deltaTime=gconst->deltaTime;
    for (int pos = tid; pos < numElements; pos += threadN)
    {
        for (int i = 0; i< numElements; i++)
        {
          if(pos!=i)
          {
               float dotV=cDistance(bot[pos].currPos,bot[i].currPos)-(bot[pos].radius+bot[i].radius);
               allconstr[array2D(pos,i, gconst->number_of_bots)].restlength=dotV;

              if(dotV>=gconst->constrLenthCreate &&
                     (allconstr[array2D(pos,i, gconst->number_of_bots)].active==true
                      or
                      allconstr[array2D(i,pos, gconst->number_of_bots)].active==true))
               {
 /*
                  if(bot[pos].typeAmount[bot[i].type]>0)
                       atomicAdd(&bot[pos].typeAmount[bot[i].type],-1);
                  if(bot[i].typeAmount[bot[pos].type]>0)
                       atomicAdd(&bot[i].typeAmount[bot[pos].type],-1);
*/

                  if(bot[pos].typeAmount[bot[i].type]>0)
                       bot[pos].typeAmount[bot[i].type]-=1;
                  if(bot[i].typeAmount[bot[pos].type]>0)
                       bot[i].typeAmount[bot[pos].type]-=1;

                  if(bot[pos].typeAmount[bot[i].type]<=0)
                  {
                       bot[pos].typeAmount[bot[i].type]=0;
                       bot[pos].link=false;
                  }
                  if(bot[i].typeAmount[bot[pos].type]<=0)
                  {
                       bot[i].typeAmount[bot[pos].type]=0;
                       bot[i].link=false;
                  }

                  if(bot[pos].typeAmountPP[bot[pos].type][bot[i].type]>0)
                       bot[pos].typeAmountPP[bot[pos].type][bot[i].type]-=1;
                  if(bot[i].typeAmountPP[bot[i].type][bot[pos].type]>0)
                       bot[i].typeAmountPP[bot[i].type][bot[pos].type]-=1;

                  if(bot[pos].typeAmountPP[bot[pos].type][bot[i].type]<=0)
                  {
                       bot[pos].typeAmountPP[bot[pos].type][bot[i].type]=0;

                  }
                  if(bot[i].typeAmountPP[bot[i].type][bot[pos].type]<=0)
                  {
                       bot[i].typeAmountPP[bot[i].type][bot[pos].type]=0;
                       bot[i].link=false;
                  }


                   allconstr[array2D(pos,i, gconst->number_of_bots)].active=false;
                   allconstr[array2D(i,pos, gconst->number_of_bots)].active=false;

               }
           }
        }
    }

}

__global__ void UpdateConstrains(Player* bot, Constraint* constr, Constraint* allconstr, GameConstants* gconst, int numElements, int numElements2)
{
    int  tid = blockIdx.x * blockDim.x + threadIdx.x;
    int  threadN = gridDim.x * blockDim.x;

    float m=10.0f;
    float deltaTime=gconst->deltaTime;

    for (int pos = tid; pos < numElements; pos += threadN)
    {

       float dotV=cDistance(bot[constr[pos].particleA].currPos, bot[constr[pos].particleB].currPos)-(bot[constr[pos].particleA].radius+bot[constr[pos].particleB].radius);
       constr[pos].restlength=dotV;
/*
       if (pos == 0)
       {
           for (int i = 0; i < 3; ++i)
              printf(" particals type %1d %5d ", i, bot[constr[pos].particleA].typeAmount[i]);

            printf("\n");
       }
*/
       if(dotV<5.5f && (Links[bot[constr[pos].particleA].type][bot[constr[pos].particleB].type]==1) &&
            bot[constr[pos].particleA].typeAmount[bot[constr[pos].particleB].type]<LinkTypeSize[bot[constr[pos].particleB].type] &&
            bot[constr[pos].particleB].typeAmount[bot[constr[pos].particleA].type]<LinkTypeSize[bot[constr[pos].particleA].type]
               )
       {

           constr[pos].active=true;
           bot[constr[pos].particleA].typeAmount[bot[constr[pos].particleB].type]+=1;
           bot[constr[pos].particleB].typeAmount[bot[constr[pos].particleA].type]+=1;
           allconstr[array2D(constr[pos].particleA,constr[pos].particleB, numElements2)].active=true;
           allconstr[array2D(constr[pos].particleB,constr[pos].particleA, numElements2)].active=true;

       }
       else
       {
           if(bot[constr[pos].particleA].typeAmount[bot[constr[pos].particleB].type]>0)
            bot[constr[pos].particleA].typeAmount[bot[constr[pos].particleB].type]-=1;
           if(bot[constr[pos].particleB].typeAmount[bot[constr[pos].particleA].type]>0)
            bot[constr[pos].particleB].typeAmount[bot[constr[pos].particleA].type]-=1;

           constr[pos].active=false;
           allconstr[array2D(constr[pos].particleA,constr[pos].particleB, numElements2)].active=false;
           allconstr[array2D(constr[pos].particleB,constr[pos].particleA, numElements2)].active=false;
       }

    }
}

__global__ void CalcConstrains(Player* bot, Constraint* constr, int numElements)
{
    int  tid = blockIdx.x * blockDim.x + threadIdx.x;
    int  threadN = gridDim.x * blockDim.x;

    float m=10.0f;
    float deltaTime=0.01f;
    float gravconst=10.0f;

    for (int pos = tid; pos < numElements; pos += threadN)
    {
       //float dotV=cDistance(bot[constr[pos].particleA].currPos, bot[constr[pos].particleB].currPos);
       float dotV=constr[pos].restlength;
       bool collide=false;
       float sumForce=0.0f;
       Vec3 directForce=Vec3(0,0,0);

       if(dotV<(bot[constr[pos].particleA].radius+bot[constr[pos].particleB].radius))
       {
           collide=true;
       }
       else
       {
           int botTypeForceFlag=1.0f;
           if(bot[constr[pos].particleA].type==bot[constr[pos].particleB].type)
               botTypeForceFlag=-1.0f;

           if(dotV>55.0f)
           {
                sumForce=-gravconst*(bot[constr[pos].particleA].mass*bot[constr[pos].particleB].mass)/(dotV*dotV);
           }
           else if(dotV<35.5f and dotV>9.5f)
           {
                sumForce=-botTypeForceFlag*gravconst*(bot[constr[pos].particleA].mass*bot[constr[pos].particleB].mass)/(dotV*dotV);
           }
           else
           {
                sumForce=gravconst*(bot[constr[pos].particleA].mass*bot[constr[pos].particleB].mass)/(dotV*dotV);
           }

       }

       if(collide)
       {
           //move to real point of collide
           float dist=0.001+(bot[constr[pos].particleA].radius+bot[constr[pos].particleB].radius)-dotV;
           float distHalf=dist/2.f;
           Vec3 directPP=bot[constr[pos].particleA].currPos-bot[constr[pos].particleB].currPos;
           bot[constr[pos].particleA].currPos=bot[constr[pos].particleA].currPos+ distHalf*cSafeNormalize(directPP,Vec3(0,0,0));
           bot[constr[pos].particleB].currPos=bot[constr[pos].particleB].currPos+ distHalf*cSafeNormalize(-directPP,Vec3(0,0,0));


           bot[constr[pos].particleA].collide=true;
           bot[constr[pos].particleB].collide=true;
           performCollision(bot[constr[pos].particleA],bot[constr[pos].particleB]);
           directForce=Vec3(0,0,0);
           sumForce=0;


       }

       Vec3 directPP=bot[constr[pos].particleA].currPos-bot[constr[pos].particleB].currPos;
       bot[constr[pos].particleA].force=bot[constr[pos].particleA].force+sumForce*cSafeNormalize(directPP,Vec3(0,0,0));
       bot[constr[pos].particleB].force=bot[constr[pos].particleB].force-sumForce*cSafeNormalize(directPP,Vec3(0,0,0));


    }
}


void initTrust(thrust::host_vector<int>& a)
{

        thrust::device_vector<int> D(10, 1);
        thrust::copy(a.begin(), a.end(), D.begin());

        // print D
        for(int i = 0; i < D.size(); i++)
            std::cout << "D[" << i << "] = " << D[i] << std::endl;


}

void initKernelParticals(Player *bots,int numElements, GameConstants* gconst)
{
    initkernelParticals<< < int(1 + numElements/ 32), 64 >> > ( bots, numElements, gconst);
    gpuAssert(hipDeviceSynchronize());
}


void updateBallVelocity(Player* bot, Constraint* allconstr,GameConstants* gconst,int numElements)
{
    UpdateBallVelocity<< < int(1 + numElements / 32), 64 >> > (bot, allconstr, gconst, numElements);
    gpuAssert(hipDeviceSynchronize());
    UpdateBallPos << < int(1 + numElements / 32), 64 >> > (bot, gconst, numElements);
    gpuAssert(hipDeviceSynchronize());
}

void updateConstrains(Player* bot, Constraint* constr, Constraint* allconstr, GameConstants* gconst, int numElements, int numElements2)
{
    UpdateConstrains2 << < int(1 + numElements/ 32), 64 >> > (bot, constr, allconstr, gconst, numElements, numElements2);
    gpuAssert(hipDeviceSynchronize());
    UpdateConstrains3 << < int(1 + numElements/ 32), 64 >> > (bot, constr, allconstr, gconst, numElements, numElements2);
    gpuAssert(hipDeviceSynchronize());

}

